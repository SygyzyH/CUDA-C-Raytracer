#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>
#include "engine/cam.cuh"
#include "engine/color.cuh"
#include "prim/sphere.cuh"
#include "prim/vec.cuh"
#include "settings.h"

// globals
// TODO: some (if not all) of these should be passed as a parameter.
uint32_t *resMat;
uint32_t *ans;
size_t pitch;
int contextSize;
int lightSize;
sphere **context;
vec **lights;
cam *camera;
// to make it faster to change the camera, a host copy is also kept
cam *cameraHost;

__global__
void renderer(uint32_t *resMat, size_t pitch, cam *camera, sphere **context, int contextSize, vec **lights, int lightSize) {
    for (int i = blockIdx.y * blockDim.y + threadIdx.y; i < HEIGHT; i += blockDim.y * gridDim.y) {
        uint32_t *resMatRow = (uint32_t *) ((char *) resMat + i * pitch);
        for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < WIDTH; j += blockDim.x * gridDim.x) {
            float x = (float) 2 * j / WIDTH - 1;
            float y = (float) 2 * i / HEIGHT - 1;

            // make ray
            // origin
            float rOrigin1, rOrigin2, rOrigin3;
            rOrigin1 = camera->origin->value[0];
            rOrigin2 = camera->origin->value[1];
            rOrigin3 = camera->origin->value[2];
            // diraction
            float rDiraction1, rDiraction2, rDiraction3;
            // calculate diraction
            float rRightScale1 = camera->right->value[0] * camera->width * x;
            float rRightScale2 = camera->right->value[1] * camera->width * x;
            float rRightScale3 = camera->right->value[2] * camera->width * x;
            float rUpScale1 = camera->up->value[0] * camera->height * y;
            float rUpScale2 = camera->up->value[1] * camera->height * y;
            float rUpScale3 = camera->up->value[2] * camera->height * y;
            float rSumUp1 = rUpScale1 + rRightScale1;
            float rSumUp2 = rUpScale2 + rRightScale2;
            float rSumUp3 = rUpScale3 + rRightScale3;
            rDiraction1 = camera->forward->value[0] + rSumUp1;
            rDiraction2 = camera->forward->value[1] + rSumUp2;
            rDiraction3 = camera->forward->value[2] + rSumUp3;
            // normalize
            float rDiractionLength = vecLengthCuda(rDiraction1, rDiraction2, rDiraction3);
            rDiraction1 /= rDiractionLength;
            rDiraction2 /= rDiractionLength;
            rDiraction3 /= rDiractionLength;

            // next, cast the ray to get the color
            float closestIntersection = INFINITY;
            float intersectingSphereX = 0;
            float intersectingSphereY = 0;
            float intersectingSphereZ = 0;
            float intersectingSphereR = 0;
            float intersectingSphereReflectivity = 0;
            uint32_t closestIntersectionC = 0;

            for (int i = 0; i < contextSize; i++) {
                // check for closest intersection
                float intersectionPoint = sphereGetIntersectionCuda(context[i], rOrigin1, rOrigin2, rOrigin3, rDiraction1, rDiraction2, rDiraction3);

                // if its the closest intersection, update
                if (intersectionPoint < closestIntersection) {
                    closestIntersection = intersectionPoint;
                    closestIntersectionC = context[i]->color;
                    intersectingSphereX = context[i]->pos->value[0];
                    intersectingSphereY = context[i]->pos->value[1];
                    intersectingSphereZ = context[i]->pos->value[2];
                    intersectingSphereR = context[i]->radius;
                    intersectingSphereReflectivity = context[i]->reflectivity;
                }
            }
            
            // once we have the closest intersection, we apply lighting, if there is an intersection
            if (closestIntersection != INFINITY) {
                for (int i = 0; i < lightSize; i++) {
                    // diffused lighting
                    // first we need the point of intersection in 3D space
                    float intersectionAt3DSpaceX = rOrigin1 + rDiraction1 * closestIntersection;
                    float intersectionAt3DSpaceY = rOrigin2 + rDiraction2 * closestIntersection;
                    float intersectionAt3DSpaceZ = rOrigin3 + rDiraction3 * closestIntersection;

                    // next, get the intersectiong point in regards to the light source
                    float intersectionPointToLight1 = lights[i]->value[0] - intersectionAt3DSpaceX;
                    float intersectionPointToLight2 = lights[i]->value[1] - intersectionAt3DSpaceY;
                    float intersectionPointToLight3 = lights[i]->value[2] - intersectionAt3DSpaceZ;
                    // normalize to make sure length stays consistant
                    float intersectionPointToLightL = vecLengthCuda(intersectionPointToLight1, intersectionPointToLight2, intersectionPointToLight3);
                    intersectionPointToLight1 /= intersectionPointToLightL;
                    intersectionPointToLight2 /= intersectionPointToLightL;
                    intersectionPointToLight3 /= intersectionPointToLightL;

                    // now we need the normal. the normal of a sphere at any point is the point itself
                    // but this must be localized and adjusted for radius
                    float sphereNormal1 = (intersectionAt3DSpaceX - intersectingSphereX) / intersectingSphereR;
                    float sphereNormal2 = (intersectionAt3DSpaceY - intersectingSphereY) / intersectingSphereR;
                    float sphereNormal3 = (intersectionAt3DSpaceZ - intersectingSphereZ) / intersectingSphereR;
                    // normalize 
                    float sphereNormalL = vecLengthCuda(sphereNormal1, sphereNormal2, sphereNormal3);
                    sphereNormal1 /= sphereNormalL;
                    sphereNormal2 /= sphereNormalL;
                    sphereNormal3 /= sphereNormalL;
                    
                    // TODO: check if anything is obscuring the light
                    // calculate diffused lighting
                    float diffused = intersectionPointToLight1 * sphereNormal1 + intersectionPointToLight2 * sphereNormal2 + intersectionPointToLight3 * sphereNormal3;
                    // clamp diffused lighting
                    diffused = max(AMBIENTILLUMINATION, min(1.0f, diffused));
                    // apply diffused lighting to color
                    closestIntersectionC = colorMult(closestIntersectionC, diffused);

                    // specular lighting
                    // get realtive camera direction to the point of intersection
                    float cameraDiraction1 = rOrigin1 - intersectionAt3DSpaceX;
                    float cameraDiraction2 = rOrigin2 - intersectionAt3DSpaceY;
                    float cameraDiraction3 = rOrigin3 - intersectionAt3DSpaceZ;
                    // normalize
                    float cameraDiractionL = vecLengthCuda(cameraDiraction1, cameraDiraction2, cameraDiraction3);
                    cameraDiraction1 /= cameraDiractionL;
                    cameraDiraction2 /= cameraDiractionL;
                    cameraDiraction3 /= cameraDiractionL;

                    // next, get the light source in regards to the intersectiong point
                    float lightToIntersectionPoint1 = intersectionAt3DSpaceX - lights[i]->value[0];
                    float lightToIntersectionPoint2 = intersectionAt3DSpaceY - lights[i]->value[1];
                    float lightToIntersectionPoint3 = intersectionAt3DSpaceZ - lights[i]->value[2];
                    // normalize to make sure length stays consistant
                    float lightToIntersectionPointL = vecLengthCuda(lightToIntersectionPoint1, lightToIntersectionPoint2, lightToIntersectionPoint3);
                    lightToIntersectionPoint1 /= lightToIntersectionPointL;
                    lightToIntersectionPoint2 /= lightToIntersectionPointL;
                    lightToIntersectionPoint3 /= lightToIntersectionPointL;
                    
                    // calculate reflection vector
                    float lightToIntersectionPointDotNorm = lightToIntersectionPoint1 * sphereNormal1 + lightToIntersectionPoint2 * sphereNormal2 + lightToIntersectionPoint3 * sphereNormal3;

                    float reflectionVector1 = lightToIntersectionPoint1 - 2 * sphereNormal1 * lightToIntersectionPointDotNorm;
                    float reflectionVector2 = lightToIntersectionPoint2 - 2 * sphereNormal2 * lightToIntersectionPointDotNorm;
                    float reflectionVector3 = lightToIntersectionPoint3 - 2 * sphereNormal3 * lightToIntersectionPointDotNorm;

                    // calculate specular lighting
                    float reflectionDotCameraDir = reflectionVector1 * cameraDiraction1 + reflectionVector2 * cameraDiraction2 + reflectionVector3 * cameraDiraction3;
                    float specularScalar = max(0.0f, min(1.0f, reflectionDotCameraDir));
                    float specular = specularScalar * specularScalar * intersectingSphereReflectivity;
                    // apply specular lighting to color
                    closestIntersectionC = colorAdd(closestIntersectionC, (uint32_t) (specular * 255));
                }
            }

            // return the closest intersection color
            resMatRow[j] = closestIntersectionC;
        }
    }
}

void RTInit() {
    // TODO: CUDA DLL's sanity check
    // define result matricies
    ans = (uint32_t *) malloc(WIDTH * HEIGHT * sizeof(uint32_t));
    hipMallocPitch(&resMat, &pitch, WIDTH * sizeof(uint32_t), HEIGHT);

    // define context
    contextSize = 4;
    sphere **contextHost = (sphere **) malloc(contextSize * sizeof(sphere *));
    
    vec *spherePos1 = buildVec(3);
    spherePos1->value[0] = 2.0f;
    spherePos1->value[1] = 0;
    spherePos1->value[2] = 0;
    vec *spherePos2 = buildVec(3);
    spherePos2->value[0] = -0.7f;
    spherePos2->value[1] = -0.1f;
    spherePos2->value[2] = 0;
    vec *spherePos3 = buildVec(3);
    spherePos3->value[0] = 2.0f;
    spherePos3->value[1] = 0;
    spherePos3->value[2] = 1.0f;
    vec *spherePos4 = buildVec(3);
    spherePos4->value[0] = 0;
    spherePos4->value[1] = 1.1f;
    spherePos4->value[2] = -2.0f;
    sphere *sphereHost1 = buildSphere(1.0f, 1.0f, buildColor(0, 0, 255), spherePos1);
    sphere *sphereHost2 = buildSphere(0.3f, 1.0f, buildColor(0, 255, 0), spherePos2);
    sphere *sphereHost3 = buildSphere(0.6f, 1.0f, buildColor(255, 0, 0), spherePos3);
    sphere *sphereHost4 = buildSphere(0.5f, 1.0f, buildColor(127, 127, 127), spherePos4);
    contextHost[0] = buildSphereCudaCopy(sphereHost1);
    contextHost[1] = buildSphereCudaCopy(sphereHost2);
    contextHost[2] = buildSphereCudaCopy(sphereHost3);
    contextHost[3] = buildSphereCudaCopy(sphereHost4);

    hipMalloc(&context, contextSize * sizeof(sphere *));
    hipMemcpy(context, contextHost, contextSize * sizeof(sphere *), hipMemcpyHostToDevice);

    freeVec(spherePos1);
    freeVec(spherePos2);
    freeVec(spherePos3);
    freeVec(spherePos4);
    free(sphereHost1);
    free(sphereHost2);
    free(sphereHost3);
    free(sphereHost4);
    free(contextHost);

    // define lighting
    lightSize = 1;
    vec **lightsHost = (vec **) malloc(lightSize * sizeof(vec *));

    vec *lightPos1 = buildVec(3);
    lightPos1->value[0] = 0.0f;
    lightPos1->value[1] = 2.0f;
    lightPos1->value[2] = 0.0f;
    lightsHost[0] = buildVecCudaCopy(lightPos1);

    hipMalloc(&lights, lightSize * sizeof(vec *));
    hipMemcpy(lights, lightsHost, lightSize * sizeof(vec *), hipMemcpyHostToDevice);

    freeVec(lightPos1);
    free(lightsHost);

    // define camera
    vec *upguide = buildVec(3);
    upguide->value[0] = 0;
    upguide->value[1] = 1;
    upguide->value[2] = 0;
    vec *target = buildVec(3);
    target->value[0] = 1.0f;
    target->value[1] = 0.0f;
    target->value[2] = 0.0f;
    vec *origin = buildVec(3);
    origin->value[0] = -2.0f;
    origin->value[1] = 0;
    origin->value[2] = 0;
    cameraHost = buildCam(FOV, screenRatio, upguide, target, origin);
    camera = buildCamCudaCopy(cameraHost);
}

void RTCleanup() {
    // free the rest of the variables used constently.
    hipFree(resMat);
    free(ans);
    freeCamCudaCopy(camera);
    freeCam(cameraHost);

    // free context
    // make a pointer of all device pointers in host
    sphere **contextHost = (sphere **) malloc(contextSize * sizeof(sphere));
    hipMemcpy(contextHost, context, contextSize * sizeof(sphere), hipMemcpyDeviceToHost);
    // free each pointer in array
    for (int i = 0; i < contextSize; i++)
        freeSphereCudaCopy(contextHost[i]);
    // free the rest of the structs
    hipFree(context);
    free(contextHost);
}

void RTTranslateCamera(float x, float y, float z) {
    camTranslate(cameraHost, x, y, z);

    // update device camera
    // can be optimized, instead of creating a new instance, simply update the existing one (if its faster)
    freeCamCudaCopy(camera);
    camera = buildCamCudaCopy(cameraHost);
}

void RTRotateCamera(float yaw, float pitch) {
    camRotate(cameraHost, yaw, pitch);

    // update device camera
    // can be optimized, instead of creating a new instance, simply update the existing one (if its faster)
    freeCamCudaCopy(camera);
    camera = buildCamCudaCopy(cameraHost);
}

uint32_t* RTEntryPoint() {
    // print some device infromation.
    // TODO: failing these runtime API calls would cause the program to crash - not the best way of testing for
    // compatible devices, but one nonetheless. see RTInit for a better solution.
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    printf("Found compatible device: %s with compute-capability %d.%d.\n", deviceProp.name, deviceProp.major, deviceProp.minor);

    // call renderer
    renderer<<<(int) (ceilf(WIDTH * HEIGHT / 1024)), 1024>>>(resMat, pitch, camera, context, contextSize, lights, lightSize);

    // get back color matrix
    // profiling has shown this is a major bottleneck, responsible for a couple of ms of delay.
    // drawing directly from the GPU would be preferrable, but would require CUDA writing to a
    // texture that is later displayed using openGL. 
    hipMemcpy2D(ans, WIDTH * sizeof(uint32_t), resMat, pitch, WIDTH * sizeof(uint32_t), HEIGHT, hipMemcpyDeviceToHost);
    
    return ans;
}
